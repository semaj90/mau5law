// Minimal CUDA test to isolate the issue
#include <hip/hip_runtime.h>

__global__ void minimal_kernel() {
    // Do nothing - just test compilation
}

extern "C" {
    void test_function() {
        // Just a test function
    }
}